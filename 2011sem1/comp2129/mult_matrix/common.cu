#include <stdio.h>
#include <stdlib.h>
#include "common.h"

float *
create_matrix_h(unsigned int w, unsigned int h) {
  float *m;
  m = (float *) malloc(w * h * sizeof(float));
  if (m == NULL) {
    fprintf(stderr, "Failed to malloc.\n");
    exit(1);
  }
  return m;
}


void
fill_matrix_h(float *const m, unsigned int w, unsigned int h, float *const values, unsigned int nvalues) {
  unsigned int i, j = 0;
  for (i = 0; i != w * h; ++i) {
    m[i] = values[j];
    j = (j + 1) % nvalues;
  }
}


void
print_matrix_h(float *const m, unsigned int w, unsigned int h, FILE *f) {
  unsigned int x, y;
  float *ptr = m;

  for (y = 0; y != h; ++y) {
    for (x = 0; x != w; ++x) 
      fprintf(f, "%.2f ", *ptr++);
    fprintf(f, "\n");
  }
}


float *
create_matrix_d(unsigned int w, unsigned int h) {
  float *m;
  if (hipMalloc(&m, w * h * sizeof(float)) == hipErrorOutOfMemory) {
    fprintf(stderr, "Failed to hipMalloc.\n");
    exit(1);
  }
  return m;
}

