#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/resource.h>
#include "common.h"

#define TILE_SIZE 16
#define SIZE (TILE_SIZE * 256)

float INIT0[] = {1.23, 23.4, 0.1, 9.12};
float INIT1[] = {213.9, 11.111, 872, -23, 16.1718};

void
cpu_mult(const float *m1, const float *m2, float *m3, unsigned int width) {
  unsigned int i, j, k;
  float result;

  for (i = 0; i != width; ++i) {
    for (j = 0; j != width; ++j) {
      result = 0;
      for (k = 0; k != width; ++k)
        result += m1[i*width + k] * m2[k*width + j];
      m3[i*width + j] = result;
    }
  }
}


__global__ void
kernel1(const float *m1, const float *m2, float *m3, unsigned int width) {
  const unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int i;
  float result = 0;

  for (i = 0; i != width; ++i) 
    result += m1[row*width + i] * m2[i*width + col];
  m3[row*width + col] = result;
}


__global__ void
kernel2(const float *m1, const float *m2, float *m3, unsigned int width) {
  const unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int t, i;
  float result = 0, a, b;

  for (t = 0; t < width / TILE_SIZE; ++t) {
    for (i = 0; i != TILE_SIZE; ++i) {
      a = m1[row*width + t*TILE_SIZE + i];
      b = m2[(t*TILE_SIZE + i)*width + col];
      result += a * b;
    }
    __syncthreads();
  }
  m3[row*width + col] = result;
}


__global__ void
kernel3(const float *m1, const float *m2, float *m3, unsigned int width) {
  __shared__ float sm1[TILE_SIZE][TILE_SIZE];
  __shared__ float sm2[TILE_SIZE][TILE_SIZE];
  const unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int t, i;
  float result = 0;

  for (t = 0; t < width / TILE_SIZE; ++t) {
    sm1[threadIdx.y][threadIdx.x] = m1[row*width + (t*TILE_SIZE + threadIdx.x)];
    sm2[threadIdx.y][threadIdx.x] = m2[(t*TILE_SIZE + threadIdx.y)*width + col];
    __syncthreads();
    
    for (i = 0; i != TILE_SIZE; ++i)
      result += sm1[threadIdx.y][i] * sm2[i][threadIdx.x];
    __syncthreads();
  }

  m3[row*width + col] = result;
}


void
print_times(const char *kernel, struct rusage *before, struct rusage *after) {
  printf("---- %s -----\n", kernel);
  printf("[sys a] sec: %ld usec: %ld\n", after->ru_stime.tv_sec, after->ru_stime.tv_usec);
  printf("[sys b] sec: %ld usec: %ld\n", before->ru_stime.tv_sec, before->ru_stime.tv_usec);
  printf("[usr a] sec: %ld usec: %ld\n", after->ru_utime.tv_sec, after->ru_utime.tv_usec);
  printf("[usr b] sec: %ld usec: %ld\n", before->ru_utime.tv_sec, before->ru_utime.tv_usec);
  printf("sys: %ld usr: %ld\n", after->ru_stime.tv_sec - before->ru_stime.tv_sec, after->ru_utime.tv_sec - before->ru_utime.tv_sec);
  printf("\n");
  fflush(stdout);
}


int
main(void) {
  float *hm[3], *dm[3];
  unsigned int i;
  struct rusage times[2];
  dim3 bdim(TILE_SIZE, TILE_SIZE);
  dim3 gdim(SIZE/TILE_SIZE, SIZE/TILE_SIZE);
  
  fprintf(stderr, "setting up matrices ... "); fflush(stderr);
  for (i = 0; i != 3; ++i) {
    hm[i] = create_matrix_h(SIZE, SIZE);
    dm[i] = create_matrix_d(SIZE, SIZE);
  }
  fill_matrix_h(hm[0], SIZE, SIZE, INIT0, sizeof(INIT0)/sizeof(float));
  fill_matrix_h(hm[1], SIZE, SIZE, INIT1, sizeof(INIT1)/sizeof(float));
  hipMemcpy(dm[0], hm[0], SIZE * SIZE * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dm[1], hm[1], SIZE * SIZE * sizeof(float), hipMemcpyHostToDevice);
  fprintf(stderr, "done\n"); fflush(stderr);

  getrusage(RUSAGE_SELF, &times[0]);
  cpu_mult(hm[0], hm[1], hm[2], SIZE);
  getrusage(RUSAGE_SELF, &times[1]);
  print_times("CPU", &times[0], &times[1]);

  getrusage(RUSAGE_SELF, &times[0]);
  kernel1<<<gdim, bdim>>>(dm[0], dm[1], dm[2], SIZE);
  hipDeviceSynchronize();
  getrusage(RUSAGE_SELF, &times[1]);
  print_times("kernel1", &times[0], &times[1]);

  getrusage(RUSAGE_SELF, &times[0]);
  kernel2<<<gdim, bdim>>>(dm[0], dm[1], dm[2], SIZE);
  hipDeviceSynchronize();
  getrusage(RUSAGE_SELF, &times[1]);
  print_times("kernel2", &times[0], &times[1]);

  getrusage(RUSAGE_SELF, &times[0]);
  kernel3<<<gdim, bdim>>>(dm[0], dm[1], dm[2], SIZE);
  hipDeviceSynchronize();
  getrusage(RUSAGE_SELF, &times[1]);
  print_times("kernel3", &times[0], &times[1]);

  for (i = 0; i != 3; ++i) {
    hipFree(dm[i]);
    free(hm[i]);
  }

  return 0;
}

