#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (32)
#define BLOCK (8)

__global__ void
matrix_add(const float *a, const float *b, float *c) {
  const int x = threadIdx.x + blockIdx.x*blockDim.x;
  const int y = threadIdx.y + blockIdx.y*blockDim.y;
  const int i = y*N + x;
  c[i] = a[i] + b[i]; 
}

void
print_matrix(const float *m, const int w, const int h) {
  int x, y;
  for (y = 0; y != h; ++y) {
    for (x = 0; x != w; ++x)
      printf("%02.0f ", m[y*w + x]);
    printf("\n");
  }
}

void
create_matrix_d(float **m, int w, int h) {
  hipMalloc(m, w * h * sizeof(float));
  hipMemset(*m, 0, w * h * sizeof(float));
}

void
create_matrix_h(float **m, int w, int h) {
  *m = (float *) malloc(w * h * sizeof(float));
  memset(*m, 0, w * h * sizeof(float));
}


int
main(void) {
  float *h_a, *h_b, *h_c;
  float *d_a, *d_b, *d_c;
  int x, y;
  
  create_matrix_h(&h_a, N, N);
  create_matrix_h(&h_b, N, N);
  create_matrix_h(&h_c, N, N);

  create_matrix_d(&d_a, N, N);
  create_matrix_d(&d_b, N, N);
  create_matrix_d(&d_c, N, N);
  
  for (y = 0; y != N; ++y) {
    for (x = 0; x != N; ++x) {
      h_a[y*N + x] = y*N + x;
      h_b[y*N + x] = 1;
    }
  }
  
  hipMemcpy(d_a, h_a, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N*N*sizeof(float), hipMemcpyHostToDevice);
  
  dim3 dimb(BLOCK, BLOCK);
  dim3 dimg(N/BLOCK, N/BLOCK);

  matrix_add<<<dimg, dimb>>>(d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, N*N*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  printf("first matrix\n");
  print_matrix(h_a, N, N);
  printf("second matrix\n");
  print_matrix(h_b, N, N);
  printf("resultant matrix\n");
  print_matrix(h_c, N, N);

  return 0;
}


