#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (32)
#define BLOCK (8)

__global__
void
matrixFill(float *matrix) {
  const int x = threadIdx.x + blockDim.x*blockIdx.x; 
  const int y = threadIdx.y + blockDim.y*blockIdx.y;
  const int i = N*y + x;
  matrix[i] = blockIdx.y;
}


int
main(void) {
  float h_matrix[N][N];
  float *d_matrix;
  unsigned int i, j;

  hipMalloc(&d_matrix, N * N * sizeof(float));
  
  dim3 dimg(N/BLOCK, N/BLOCK);
  dim3 dimb(BLOCK, BLOCK);
  matrixFill<<<dimg, dimb>>>(d_matrix);

  hipMemcpy(h_matrix, d_matrix, N * N * sizeof(float), hipMemcpyDeviceToHost);
  
  for (i = 0; i != N; ++i) {
    for (j = 0; j != N; ++j)
      printf("%02.0f ", h_matrix[j][i]);
    printf("\n");
  }

  return 0;
}
