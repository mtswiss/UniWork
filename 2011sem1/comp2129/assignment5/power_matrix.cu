#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void
matrix_multipy(float *M, float *I, float *R, int w){
	int x = threadIdx.x; 
	int y = threadIdx.y;
	float Rvalue = 0;
	for(int i = 0; i< w;i++){
		Rvalue += M[y*w + i] * I[i*w + x];
	}
	R[y*w + x] = Rvalue;
}

int
main(void) {
	int w, power;
	
	if(scanf("%d\n", &w) != 1 || w < 1){
		printf("error\n");
		return -1;
	}

	if(scanf("%d\n", &power) != 1 || power < 0){
		printf("error\n");
		return -1;
	}
	

	float *M = (float *)malloc(w * w * sizeof(float));
	float *I = (float *)malloc(w * w * sizeof(float));
	float *R = (float *)malloc(w * w * sizeof(float));

	float *dM, *dI, *dR;
	hipMalloc(&dM, w*w*sizeof(float));
	hipMalloc(&dI, w*w*sizeof(float));
	hipMalloc(&dR, w*w*sizeof(float));

	
	if(power==0){
		int j, k =0;
		for(j = 0; j<w;j++){
			for(k = 0; k<w;k++){
				if(j==k)
					printf("%.3f ",1.0);
				else
					printf("%.3f ",0.0);
			}

			printf("\n");
		}

		free(M);
		free(I);
		free(R);
		return -1;
	}
			
	
	for(int i = 0; i < w*w; i++){
		float x;
		if(scanf("%f",&x) == 1){
			M[i] = x;
			I[i] = x;
		}
		else {
			printf("error\n");
			return -1;
		}
	}
	
	hipMemcpy(dM, M, w*w*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dI, I, w*w*sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock (w,w);
	dim3 dimGrid(1);

	for(int i = 1; i<power; i++){
		matrix_multipy<<<dimGrid,dimBlock>>>(dM,dI,dR,w);
		hipDeviceSynchronize();
		hipMemcpy(dI, dR, w*w*sizeof(float), hipMemcpyDeviceToHost);
	}

	hipMemcpy(R,dR,w*w*sizeof(float),hipMemcpyDeviceToHost);
	
	for (int i = 0; i < w * w; i++) {
		if (power == 1)
			printf("%.3f ", M[i]);
		else
			printf("%.3f ", R[i]);
		if ((i + 1) % w == 0)
			printf("\n");
	}
	
	hipFree(dR);
	hipFree(dI);
	hipFree(dR);
	free(M);
	free(I);
	free(R);
	
  return 0;
}

